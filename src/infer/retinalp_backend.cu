#include "hip/hip_runtime.h"


#include "retinalp_backend.hpp"
#include <cc_util.hpp>
#include <common/trt_common.hpp>

namespace TRTInfer {

	RetinaLPBackend::RetinaLPBackend(int max_objs, CUStream stream) :Backend(stream) {
		this->max_objs_ = max_objs;
	}

	static __global__ void RetinaLPBackend_forwardGPU(float* conf, float* offset, float* landmark, float *anchors_matrix,
		int* countptr, ccutil::LPRBox* boxptr, float threshold, int maxobjs, int edge) {

		KERNEL_POSITION;

		float confidence = conf[position * 2 + 1];
		if (confidence < threshold)
			return;
		int index = atomicAdd(countptr, 1);
		if (index >= maxobjs)
			return;

		float cx_a = anchors_matrix[position * 4];
		float cy_a = anchors_matrix[position * 4 + 1];
		float w_a = anchors_matrix[position * 4 + 2];
		float h_a = anchors_matrix[position * 4 + 3];
		//printf("anchors_matrix:%f,%f,%f,%f \n", cx_a, cy_a, w_a, h_a);
		float loc_x = offset[position * 4];
		float loc_y = offset[position * 4 + 1];
		float loc_w = offset[position * 4 + 2];
		float loc_h = offset[position * 4 + 3];
		float cx_b = cx_a + loc_x * 0.1 * w_a;
		float cy_b = cy_a + loc_y * 0.1 * h_a;
		float w_b = w_a * expf(loc_w * 0.2);
		float h_b = h_a * expf(loc_h * 0.2);

		ccutil::LPRBox* ptr = boxptr + index;
		ptr->x = cx_b - w_b * 0.5;
		ptr->y = cy_b - h_b * 0.5;
		ptr->r = cx_b + w_b * 0.5;
		ptr->b = cy_b + h_b * 0.5;
		ptr->score = confidence;
		ptr->label = 0;

		for (int k = 0; k < 4; ++k) {
			float landmark_x = cx_a + 0.1 * landmark[position *8 + k * 2] * w_a;
			float landmark_y = cy_a + 0.1 * landmark[position *8 + k * 2 + 1] * h_a;

			cv::Point2f& point = ptr->landmark[k];
			point.x = landmark_x;
			point.y = landmark_y;
		}
	}

	void RetinaLPBackend::postProcessGPU(shared_ptr<Tensor> conf, shared_ptr<Tensor> offset,
		shared_ptr<Tensor> landmark, Mat anchors_matrix, 
		float threshold, vector<vector<ccutil::LPRBox>> &bboxs) {

		int batchSize = conf->num();
		int total_pix = conf->height();
		int job_count = total_pix;
		auto grid = gridDims(job_count);
		auto block = blockDims(job_count);
		size_t objsStoreSize = max_objs_ * sizeof(ccutil::LPRBox) + sizeof(int);

		void* cpuPtr = getCPUMemory(objsStoreSize * batchSize);
		char* cpuPtrInput = (char*)cpuPtr;
		void* gpuPtr = getGPUMemory(objsStoreSize * batchSize);
		char* gpuPtrInput = (char*)gpuPtr;

		void* gpu_anchors_Ptr = getGPUMemory(total_pix * 4 * sizeof(float));
		float* gpu_anchors_matrix_ = (float*)gpu_anchors_Ptr;

		auto stream = getStream();
		float* anchors_matrix_ptr = anchors_matrix.ptr<float>(0);
		hipMemcpyAsync(gpu_anchors_matrix_, anchors_matrix_ptr, total_pix * 4 * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice, stream);

		for (int i = 0; i < batchSize; ++i) {

			float* conf_ptr = conf->gpu<float>(i);
			float* offset_ptr = offset->gpu<float>(i);
			float* landmark_ptr = landmark->gpu<float>(i);
			int* countPtr = (int*)gpuPtrInput;
			ccutil::LPRBox* boxPtr = (ccutil::LPRBox*)((char*)gpuPtrInput + sizeof(int));
			hipMemsetAsync(gpuPtrInput, 0, sizeof(int), stream);
			RetinaLPBackend_forwardGPU << < grid, block, 0, stream >> > (conf_ptr, offset_ptr, landmark_ptr, gpu_anchors_matrix_,
				countPtr, boxPtr, threshold, max_objs_, job_count);

			hipMemcpyAsync(cpuPtrInput, gpuPtrInput, objsStoreSize, hipMemcpyKind::hipMemcpyDeviceToHost, stream);
			cpuPtrInput += objsStoreSize;
			gpuPtrInput += objsStoreSize;
		}
		hipStreamSynchronize(stream);

		cpuPtrInput = (char*)cpuPtr;
		for (int n = 0; n < batchSize; ++n, cpuPtrInput += objsStoreSize) {
			auto& output = bboxs[n];
			output.clear();
			int num = *((int*)cpuPtrInput);
			num = std::min(num, max_objs_);
			if (num == 0)
				continue;

			ccutil::LPRBox* ptr = (ccutil::LPRBox*)(cpuPtrInput + sizeof(int));
			output.insert(output.begin(), ptr, ptr + num);
		}
	}
};