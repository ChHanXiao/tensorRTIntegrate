#include "hip/hip_runtime.h"


#include "yolov5_detect_backend.hpp"
#include <cc_util.hpp>
#include <common/trt_common.hpp>

namespace TRTInfer {

	YOLOv5DetectBackend::YOLOv5DetectBackend(int max_objs, CUStream stream) :Backend(stream) {

		this->max_objs_ = max_objs;
	}

	static __device__ float sigmoid(float value) {
		return 1 / (1 + exp(-value));
	}

	static float desigmoid(float val) {
		return -log(1 / val - 1);
	}

	struct Anchor {
		int width[9], height[9];
	};

	static __global__ void decode_native_impl(float* data,
		int width, int height, int stride, float threshold, float threshold_desigmoid, int num_classes,
		Anchor anchor, ccutil::BBox* output, int* counter, int area, int maxobjs, int edge) {

		KERNEL_POSITION;

		int inner_offset = position % area;
		int a = position / area;
		float* ptr = data + (a * (num_classes + 5) + 4) * area + inner_offset;

		if (*ptr < threshold_desigmoid)
			return;

		float obj_confidence = sigmoid(*ptr);
		float* pclasses = ptr + area;
		float max_class_confidence = *pclasses;
		int max_classes = 0;
		pclasses += area;

		for (int j = 1; j < num_classes; ++j, pclasses += area) {
			if (*pclasses > max_class_confidence) {
				max_classes = j;
				max_class_confidence = *pclasses;
			}
		}

		max_class_confidence = sigmoid(max_class_confidence) * obj_confidence;
		if (max_class_confidence < threshold)
			return;

		int index = atomicAdd(counter, 1);
		if (index >= maxobjs)
			return;

		float* pbbox = ptr - 4 * area;
		float dx = sigmoid(*pbbox);  pbbox += area;
		float dy = sigmoid(*pbbox);  pbbox += area;
		float dw = sigmoid(*pbbox);  pbbox += area;
		float dh = sigmoid(*pbbox);  pbbox += area;

		int cell_x = position % width;
		int cell_y = (position / width) % height;
		float cx = (dx * 2 - 0.5f + cell_x) * stride;
		float cy = (dy * 2 - 0.5f + cell_y) * stride;
		float w = pow(dw * 2, 2) * anchor.width[a];
		float h = pow(dh * 2, 2) * anchor.height[a];
		float x = cx - w * 0.5f;
		float y = cy - h * 0.5f;
		float r = cx + w * 0.5f;
		float b = cy + h * 0.5f;
		ccutil::BBox& box = output[index];
		box.x = x;
		box.y = y;
		box.r = r;
		box.b = b;
		box.label = max_classes;
		box.score = max_class_confidence;
	}


	void YOLOv5DetectBackend::postProcessGPU(shared_ptr<Tensor> tensor, int stride, float threshold, int num_classes,
		const vector<vector<float>>& anchors, vector<vector<ccutil::BBox>> &bboxs) {

		float threshold_desigmoid = desigmoid(threshold);
		int tensor_width = tensor->width();
		int tensor_height = tensor->height();
		int batchSize = tensor->num();
		size_t objsStoreSize = max_objs_ * sizeof(ccutil::BBox) + sizeof(int);
		int area = tensor_width * tensor_height;
		void* cpuPtr = getCPUMemory(objsStoreSize * batchSize);
		char* cpuPtrInput = (char*)cpuPtr;
		void* gpuPtr = getGPUMemory(objsStoreSize * batchSize);
		char* gpuPtrInput = (char*)gpuPtr;
		auto stream = getStream();
		int job_count = area * anchors.size();
		auto grid = gridDims(job_count);
		auto block = blockDims(job_count);
		Anchor anchor;

		for (int i = 0; i < anchors.size(); ++i) {
			anchor.width[i] = anchors[i][0];
			anchor.height[i] = anchors[i][1];
		}

		for (int n = 0; n < batchSize; ++n) {

			int* counter = (int*)gpuPtrInput;
			ccutil::BBox* bboxptr = (ccutil::BBox*)((char*)gpuPtrInput + sizeof(int));

			hipMemsetAsync(counter, 0, sizeof(int), stream);
			decode_native_impl << < grid, block, 0, stream >> > (
				tensor->gpu<float>(n),
				tensor_width, tensor_height, stride, threshold, threshold_desigmoid, num_classes,
				anchor, bboxptr, counter, area, max_objs_, job_count);

			hipMemcpyAsync(cpuPtrInput, gpuPtrInput, objsStoreSize, hipMemcpyKind::hipMemcpyDeviceToHost, stream);

			cpuPtrInput += objsStoreSize;
			gpuPtrInput += objsStoreSize;
		}
		hipStreamSynchronize(stream);

		cpuPtrInput = (char*)cpuPtr;
		for (int n = 0; n < batchSize; ++n, cpuPtrInput += objsStoreSize) {
			auto& output = bboxs[n];

			int num = *((int*)cpuPtrInput);
			num = std::min(num, max_objs_);
			if (num == 0)
				continue;

			ccutil::BBox* ptr = (ccutil::BBox*)(cpuPtrInput + sizeof(int));
			output.insert(output.begin(), ptr, ptr + num);
		}
	}
};